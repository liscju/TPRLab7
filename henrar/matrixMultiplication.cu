#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

#include <cstdio>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#define BLOCK_SIZE 16

class Matrix
{
public:	
	int width;
	int height;
	float *elements;
};

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	StopWatchInterface *timer = NULL;
	float elapsedTime = 0.0f;
	hipEvent_t start, stop;
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
        dim3 dimBlock(128); // threads per block?
        dim3 dimGrid(128); // number of blocks?


	sdkCreateTimer(&timer);
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));


	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipDeviceSynchronize());
	sdkStopTimer(&timer);
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "Time: " << elapsedTime << std::endl;
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{	
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > A.height || col > B.width)
	{
		return;
	}
	for (int e = 0; e < A.width; ++e)
	{
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	}
	C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char * const argv[])
{	
	int Width = 16;
	
	Matrix A;
	Matrix B;
	Matrix C;
	
	A.width = Width;
	B.width = Width;
	C.width = Width;
	
	A.height = Width;
	B.height = Width;
	C.height = Width;
	
	A.elements = new float[Width*Width];
	B.elements = new float[Width*Width];
	C.elements = new float[Width*Width];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open("input/A.txt");
	B_input.open("input/B.txt");
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;
	while (!A_input.eof())
	{	A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}
	A_input.close();
	B_input.close();

	MatMul(A, B, C);
	std::ofstream C_output;
	C_output.open("output/gpu_results.txt");
	for (int i = 0; i < Width; i++)
	{	for (int j = 0; j < Width; j++)
		{
			C_output << C.elements[i*Width+j]<<"\t";
		}
		C_output << std::endl;
	}
	
	C_output.close();
	hipDeviceReset();
	return 0;
}
	
