#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;
#include <stdio.h>
#include <iostream>
#include <fstream>

#include <hip/hip_runtime.h>
#include "helper_functions.h"

#define BLOCK_SIZE 16
#define ITER_COUNT 1000

typedef struct {
	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMulGPU(const Matrix A, const Matrix B, Matrix C) {
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
    dim3 dimBlock(256); // threads per block?
    dim3 dimGrid(256); // number of blocks?

	// TIMER START
	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	for(int i=0; i<ITER_COUNT; ++i) {
		MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	}

	// TIMER STOP
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	printf("GPU Elapsed time: %f\n", time/ITER_COUNT);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {	
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row > A.height || col > B.width) return;
	for (int e = 0; e < A.width; ++e) {
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	}
	C.elements[row * C.width + col] = Cvalue;
}

void MatMulCPU(Matrix A, Matrix B, Matrix C) {
	// TIMER START
	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	for(int i=0; i<ITER_COUNT; ++i) {		
		for(int row=0; row<A.height; row++) {
			for(int col=0; col<B.width; col++) {
				float Cvalue = 0;
				for (int e = 0; e < A.width; ++e) {
					Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
				}
				C.elements[row * C.width + col] = Cvalue;
			}
		}
	}

	// TIMER STOP
	sdkStopTimer(&timer);
	float time = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	printf("CPU Elapsed time: %f\n", time/ITER_COUNT);
}

int main(int argc, char * const argv[]) {	
	int Width = 16;
	
	Matrix A;
	Matrix B;
	Matrix C;
	
	A.width = Width;
	B.width = Width;
	C.width = Width;
	
	A.height = Width;
	B.height = Width;
	C.height = Width;
	
	A.elements = new float[Width*Width];
	B.elements = new float[Width*Width];
	C.elements = new float[Width*Width];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open("A.txt");
	B_input.open("B.txt");
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;
	while (!A_input.eof()) {
		A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}
	A_input.close();
	B_input.close();

	// TIMER START
	StopWatchInterface *timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	MatMulGPU(A, B, C);

	// TIMER STOP
	sdkStopTimer(&timer);
	float gpuTime = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);

	
	// TIMER START
	timer=NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	MatMulCPU(A, B, C);

	// TIMER STOP
	sdkStopTimer(&timer);
	float cpuTime = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);


	std::ofstream C_output;
	C_output.open("C.txt");
	for (int i=0; i<Width; i++) {
		for (int j=0; j<Width; j++) {
			C_output<<C.elements[i*Width+j]<<"\t";
		}
		C_output<<endl;
	}
	C_output.close();

	printf("GPU Elapsed total time: %f\n", gpuTime);
	printf("CPU Elapsed total time: %f\n", cpuTime);

	delete A.elements;
	delete B.elements;
	delete C.elements;

	hipDeviceReset();
	return 0;
}
	
